///////////////////////////////////////////////////////////////////////////
/// PROGRAMACIÓN EN CUDA C/C++
/// Práctica:	BASICO 1 : Memoria Global
/// Autor:		Gustavo Gutierrez Martin
/// Fecha:		Septiembre 2022
///////////////////////////////////////////////////////////////////////////

/// dependencias ///

#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>

/// constantes ///
#define MB (1<<20) /// MiB = 2^20
#define N 16 /// Tamaño del array de datos

/// definición de funciones ///

/// muestra por consola que no se ha encontrado un dispositivo CUDA
int getErrorDevice();

/// muestra los datos de los dispositivos CUDA encontrados
int getDataDevice(int deviceCount);

/// numero de CUDA cores
int getCudaCores(hipDeviceProp_t deviceProperties);

/// muestra por pantalla las propiedades del dispositivo CUDA
int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties);

/// función que muestra por pantalla la salida del programa
int getAppOutput();

/// muestra por pantalla los datos del host
int printHostData(float *hst_A, float *hst_B);

/// inicializa el array del host
int loadHostData(float *hst_A, float *hst_B);

/// transfiere los datos
int dataTransfer(float *hst_A, float *hst_B,float *dev_A, float *dev_B );

/// función principal de la aplicación
int main() {
    /// declaración de variables
    int deviceCount;
    float *hst_A,*hst_B,*dev_A,*dev_B;

    /// reserva del espacio de memoria en el host
    hst_A = (float*)malloc( N * sizeof(float) );
    hst_B = (float*)malloc( N * sizeof(float) );
    /// reserva del espacio de memoria en el device
    hipMalloc( (void**)&dev_A, N * sizeof(float) );
    hipMalloc( (void**)&dev_B, N * sizeof(float) );

    /// cargamos los datos iniciales en el host
    loadHostData(hst_A, hst_B);

    /// transferimos los datos
    dataTransfer(hst_A, hst_B, dev_A, dev_B);

    /// buscando dispositivos
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        /// mostramos el error si no se encuentra un dispositivo
        return getErrorDevice();
    } else {
        /// mostramos los datos de los dispositivos CUDA encontrados
        getDataDevice(deviceCount);
    }
    /// mostramos por pantalla los datos del host
    printHostData(hst_A, hst_B);

    /// liberamos los recursos del device
    hipFree(dev_A);
    hipFree(dev_B);

    /// mostramos el final del programa
    getAppOutput();
    return 0;
}

int getErrorDevice() {
    printf("¡No se ha encontrado un dispositivo CUDA!\n");
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 1;
}

int getDataDevice(int deviceCount) {
    printf("Se han encontrado %d dispositivos CUDA:\n", deviceCount);
    for (int deviceID = 0; deviceID < deviceCount; deviceID++) {
        ///obtenemos las propiedades del dispositivo CUDA
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, deviceID);
        getDeviceProperties(deviceID, getCudaCores(deviceProp), deviceProp);
    }
    return 0;
}

int getCudaCores(hipDeviceProp_t deviceProperties) {
    int cudaCores = 0;
    int major = deviceProperties.major;
    if (major == 1) {
        /// TESLA
        cudaCores = 8;
    } else if (major == 2) {
        /// FERMI
        if (deviceProperties.minor == 0) {
            cudaCores = 32;
        } else {
            cudaCores = 48;
        }
    } else if (major == 3) {
        /// KEPLER
        cudaCores = 192;
    } else if (major == 5) {
        /// MAXWELL
        cudaCores = 128;
    } else if (major == 6 || major == 7 || major == 8) {
        /// PASCAL, VOLTA (7.0), TURING (7.5), AMPERE
        cudaCores = 64;
    } else {
        /// ARQUITECTURA DESCONOCIDA
        cudaCores = 0;
        printf("¡Dispositivo desconocido!\n");
    }
    return cudaCores;
}

int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties) {
    int SM = cudaProperties.multiProcessorCount;
    printf("***************************************************\n");
    printf("DEVICE %d: %s\n", deviceId, cudaProperties.name);
    printf("***************************************************\n");
    printf("- Capacidad de Computo            \t: %d.%d\n", cudaProperties.major, cudaProperties.minor);
    printf("- No. de MultiProcesadores        \t: %d \n", SM);
    printf("- No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores * SM);
    printf("- Memoria Global (total)          \t: %zu MiB\n", cudaProperties.totalGlobalMem / MB);
    printf("***************************************************\n");
    return 0;
}

int getAppOutput() {
    /// salida del programa
    time_t fecha;
    time(&fecha);
    printf("***************************************************\n");
    printf("Programa ejecutado el: %s", ctime(&fecha));
    printf("***************************************************\n");
    /// capturamos un INTRO para que no se cierre la consola de MSVS
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}

int printHostData(float *hst_A, float *hst_B) {
    printf("ENTRADA:\n");
    for (int i=0; i<N; i++)  {
        printf("HST_A[%i] = %.2f\n", i, hst_A[i]);
    }
    printf("\n");
    printf("SALIDA:\n");
    for (int i=0; i<N; i++)  {
        printf("HST_B[%i] = %.2f\n", i, hst_B[i]);
    }
    printf("\n");
    return 0;
}

int loadHostData(float *hst_A, float *hst_B) {
    srand ( (int)time(nullptr) );
    for (int i=0; i<N; i++)  {
        /// inicializamos hst_A con numeros aleatorios entre 0 y 1
        hst_A[i] = (float) rand() / RAND_MAX;
        /// inicializamos hst_B con ceros
        hst_B[i] = 0;
    }
    return 0;
}

int dataTransfer(float *hst_A, float *hst_B,float *dev_A, float *dev_B ) {
    /// transfiere datos de hst_A a dev_A
    hipMemcpy(dev_A,hst_A,N * sizeof(float),hipMemcpyHostToDevice);
    /// transfiere datos de dev_A a dev_B
    hipMemcpy(dev_B,dev_A,N * sizeof(float),hipMemcpyDeviceToDevice);
    /// transfiere datos de dev_B a hst_B
    hipMemcpy(hst_B,dev_B,N * sizeof(float),hipMemcpyDeviceToHost);
    return 0;
}
